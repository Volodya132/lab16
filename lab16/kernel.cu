#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void compute(double a, double b, double c, double e, double f, double g, double h, double* res)
{
    *res = a+b+c+e+f+g/h;
}

int main()
{
    double cpu_res;
    double* gpu_res;
    
    hipMalloc((void**)&gpu_res, sizeof(double));
    compute << <1, 1>>>(1, 2, 3, 4, 5, 8., 3., gpu_res);
    hipMemcpy(&cpu_res, gpu_res, sizeof(double), hipMemcpyDeviceToHost);
    printf("1+2+3+4+5+8/3=%lf\n", cpu_res);
    hipFree(gpu_res);



    return 0;
}

